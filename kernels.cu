#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

typedef char DensityType;
typedef float4 VelocityType;

hipArray* d_densityPingArray = 0;
hipArray* d_densityPongArray = 0;
hipArray* d_velocityPingArray = 0;
hipArray* d_velocityPongArray = 0;

DensityType* d_densityPing;
DensityType* d_densityPong;
VelocityType* d_velocityPing;
VelocityType* d_velocityPong;

texture<DensityType, 3, hipReadModeNormalizedFloat> densityTex; // 3D texture
texture<VelocityType, 3, hipReadModeElementType> velocityTex; // 3D texture

hipExtent volumeSize2;

extern "C" void checkCudaErr( const char* msg );
extern "C" void updateDensityTex(void* src);

__global__
void writeToPboKernel( char* pbo, DensityType* d_density, uint width, uint height, uint depth, int magic ) {
	int i = blockIdx.x % magic * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x / magic * blockDim.z + threadIdx.z;

	if (i < width && j < height && k < depth) {
		int idx = (k * (height * width) + j * width + i);
		pbo[idx] = d_density[idx];
	}
}
__global__
void diffuseDensityKernel( DensityType* data, DensityType* data0, float diff, float dt, int domX, int domY, int domZ, int magic ) {

	int i = blockIdx.x % magic * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x / magic * blockDim.z + threadIdx.z;

	if (i > 1 && i < domX - 1 && j > 1 && j < domY - 1 && k > 1 && k < domZ - 1) {
		for (int iter = 0; iter < 20; iter++) {
			char tmp = (data0[k * (domY * domX) + j * domX + i] + diff * (data0[k * (domY * domX) + j * domX + (i - 1)] + data0[k * (domY * domX) + j * domX + (i + 1)] + data0[k
					* (domY * domX) + (j - 1) * domX + i] + data0[k * (domY * domX) + (j + 1) * domX + i] + data0[(k - 1) * (domY * domX) + j * domX + i] + data0[(k + 1) * (domY
					* domX) + j * domX + (i + 1)])) / (1 + 6 * diff);

			data[k * (domY * domX) + j * domX + i] = min(250, tmp);
		}
		//data[k * (domY * domX) + j * domX + i] = 200;
	} else {
		data[k * (domY * domX) + j * domX + i] = 0;
	}
}


__global__
void advectDenistyKernel( DensityType* data, VelocityType* velocity, float dt, int domX, int domY, int domZ, int magic ) {

	int i = blockIdx.x % magic * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k = blockIdx.x / magic * blockDim.z + threadIdx.z;

	if (i < domX && j < domY && k < domZ) {

/*
		float x = i - dt * velocity[k * (domY * domX) + j * domX + i].x;
		float y = j - dt * velocity[k * (domY * domX) + j * domX + i].y;
		float z = k - dt * velocity[k * (domY * domX) + j * domX + i].z;
		*/
		float x = i - dt * 0;
		float y = j - dt * 0;
		float z = k - dt * 0;

		float sample = tex3D(densityTex, x, y, z);
		data[k * (domY * domX) + j * domX + i] = (DensityType)(sample * 255);

	}
}

extern "C" void initCfd( char* h_volume, void* h_velocity, hipExtent volumeSize ) {

	volumeSize2 = volumeSize;


	size_t size = volumeSize.height * volumeSize.width * volumeSize.depth;

	hipMalloc((void**) &d_densityPing, size * sizeof(DensityType));
	hipMalloc((void**) &d_densityPong, size * sizeof(DensityType));
	hipMalloc((void**) &d_velocityPing, size * sizeof(VelocityType));
	hipMalloc((void**) &d_velocityPong, size * sizeof(VelocityType));

	hipMemcpy(d_densityPing, h_volume, size * sizeof(DensityType), hipMemcpyHostToDevice);
	hipMemcpy(d_densityPong, h_volume, size * sizeof(DensityType), hipMemcpyHostToDevice);
	hipMemcpy(d_velocityPing, h_velocity, size * sizeof(VelocityType), hipMemcpyHostToDevice);
	hipMemcpy(d_velocityPong, h_velocity, size * sizeof(VelocityType), hipMemcpyHostToDevice);

	/** tex bussiness */
	hipChannelFormatDesc densityChannelDesc = hipCreateChannelDesc<DensityType> ();
	hipMalloc3DArray(&d_densityPingArray, &densityChannelDesc, volumeSize);
	hipMalloc3DArray(&d_densityPongArray, &densityChannelDesc, volumeSize);

	hipChannelFormatDesc velocityChannelDesc = hipCreateChannelDesc<VelocityType> ();
	hipMalloc3DArray(&d_velocityPingArray, &velocityChannelDesc, volumeSize);
	hipMalloc3DArray(&d_velocityPongArray, &velocityChannelDesc, volumeSize);

	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(h_volume, volumeSize.width * sizeof(DensityType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_densityPingArray;
	copyParams.extent = volumeSize;
	copyParams.kind = hipMemcpyHostToDevice;
	hipMemcpy3D(&copyParams);

	copyParams.dstArray = d_densityPongArray;
	//hipMemcpy3D(&copyParams);

	copyParams.srcPtr = make_hipPitchedPtr(h_velocity, volumeSize.width * sizeof(VelocityType), volumeSize.width, volumeSize.height);
	copyParams.dstArray = d_velocityPingArray;
	hipMemcpy3D(&copyParams);

	copyParams.dstArray = d_velocityPongArray;
	//hipMemcpy3D(&copyParams);

	checkCudaErr("Kopiranje arrayev");

	densityTex.filterMode = hipFilterModeLinear;
	velocityTex.filterMode = hipFilterModeLinear;

	hipBindTextureToArray(densityTex, d_densityPingArray, densityChannelDesc);
	hipBindTextureToArray(velocityTex, d_velocityPingArray, velocityChannelDesc);

	checkCudaErr("Bindanje textur");

}

extern "C" void simulate( dim3 gridSize, dim3 blockSize, char* pbo, hipExtent volumeSize, int magic ) {


	float dt = 0.05f;

	// delamo diffuse v pong rezultat
	diffuseDensityKernel<<<gridSize, blockSize>>>( d_densityPing, d_densityPong, 0.10f, dt, volumeSize.width, volumeSize.height, volumeSize.depth, magic);
	updateDensityTex(d_densityPing); // iz ping arraya v ping teksturo
/*
	// pingpong
	DensityType* tmp = d_densityPing;
	d_densityPing = d_densityPong;
	d_densityPong = tmp;
*/
	advectDenistyKernel<<<gridSize, blockSize>>>(d_densityPing, d_velocityPing, dt, volumeSize.width, volumeSize.height, volumeSize.depth, magic );
	//updateDensityTex(d_densityPing); // iz ping arraya v ping teksturo
	writeToPboKernel<<<gridSize, blockSize>>>(pbo, d_densityPing, volumeSize.width, volumeSize.height, volumeSize.depth, magic);


	// pingpong
	DensityType* tmp = d_densityPing;
	d_densityPing = d_densityPong;
	d_densityPong = tmp;

	hipDeviceSynchronize();
	checkCudaErr("writeToPboKernel");
}

extern "C" void updateDensityTex(void* src) {
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<DensityType> ();

	// copy data to 3D array
	hipMemcpy3DParms copyParams = { 0 };
	copyParams.srcPtr = make_hipPitchedPtr(src, volumeSize2.width * sizeof(DensityType), volumeSize2.width, volumeSize2.height);
	copyParams.dstArray = d_densityPingArray;
	copyParams.extent = volumeSize2;
	copyParams.kind = hipMemcpyDeviceToDevice;
	hipMemcpy3D(&copyParams);
	checkCudaErr("updateDensityTex failed");
}

extern "C" void checkCudaErr( const char* msg ) {
	hipError_t err = hipGetLastError();

	if (err != hipSuccess) {
		fprintf(stderr, "\nCUDA error: %s (%s)\n", msg, hipGetErrorString(err));
	}
}
